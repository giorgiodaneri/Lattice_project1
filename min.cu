#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
// extern "C++"
// {
// #include "../../include/CGSolver.hpp"
// #include "../../include/CGSolverCuda.hpp"
// }


void kernel_wrapper(std::vector<int> arr)
{
    int *d_arr;
    hipMalloc(&d_arr, arr.size() * sizeof(int));
    hipMemcpy(d_arr, arr.data(), arr.size() * sizeof(int), hipMemcpyHostToDevice);
    // CGSolverCuda solver;
    // solver.solve(d_arr, arr.size());
    hipFree(d_arr);
}