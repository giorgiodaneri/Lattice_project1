#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <random>
#include <iostream>
#include <ctime>
#include <omp.h>

#define BLOCK_SIZE 1024
#define GRID_SIZE 1024

// write a kernel that computes the minimum of an array of integers
__global__ void findMinFixpointKernel(int *arr, int *size, int *min) {
    // global thread identifier
    unsigned int unique_id = blockIdx.x * blockDim.x + threadIdx.x;
    // thread identifier within the block (used to access share memory)
    unsigned int thread_id = threadIdx.x;
    // declare chunk of shared memory
    __shared__ int minChunk[BLOCK_SIZE];

    // load elements into shared memory only if within bounds
    if (unique_id < *size) {
        minChunk[thread_id] = arr[unique_id];
    } else {
        // make sure that values out of bounds are set to a large value
        minChunk[thread_id] = INT_MAX;  
    }

    // update the global minimum if a smaller value is found within the block
    if(thread_id < *size && minChunk[thread_id] < *min) {
        *min = minChunk[thread_id];
    }
}

__global__ void findMinKernel(int *arr, int *size, int *min) {
    unsigned int unique_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_id = threadIdx.x;
    __shared__ int minChunk[BLOCK_SIZE];

    // load elements into shared memory only if within bounds
    if (unique_id < *size) {
        minChunk[thread_id] = arr[unique_id];
    } else {
        // make sure that values out of bounds are set to a large value
        minChunk[thread_id] = INT_MAX;  
    }

    __syncthreads();

    // perform reduction to find the minimum in the current block
    // this has complexity O(log(n)) where n is the number of elements in the block
    // since the reduction pattern amounts to organizing the elements in a binary tree
    // the stride is reduced by half at each iteration, and memory accesses are more coalesced
    // at the later iterations
    # pragma unroll
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (thread_id < s) {
            if (minChunk[thread_id] > minChunk[thread_id + s]) {
                minChunk[thread_id] = minChunk[thread_id + s];
            }
        }
        __syncthreads();
    }

    // perform atomic update of global minimum from local minima of each block
    // which is found in the first element of the shared memory (after reduction)
    if (thread_id == 0) {
        atomicMin(min, minChunk[0]);
    }
}


void kernel_wrapper(std::vector<int> &arr)
{
    int *d_arr;
    int *d_min;
    int *d_size;
    int min_value = 1000;
    int prev_min_value = 1001;
    int iters = 0;
    int size = arr.size();

    // measure time for memory allocation and data transfer
    hipError_t err;
    time_t start_mem, end_mem;
    start_mem = clock();
    // initialize cuda context
    hipFree(0);
    // allocate memory on the device
    err = hipMalloc((void **)&d_arr, arr.size() * sizeof(int));
    if(hipGetLastError() != hipSuccess) {
        printf("Error0: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **)&d_min, sizeof(int));
    if(hipGetLastError() != hipSuccess) {
        printf("Error1: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **)&d_size, sizeof(int));
    if(hipGetLastError() != hipSuccess) {
        printf("Error2: %s\n", hipGetErrorString(err));
    }
    // initialize all the device variables
    err = hipMemcpy(d_arr, arr.data(), arr.size() * sizeof(int), hipMemcpyHostToDevice);
    if(hipGetLastError() != hipSuccess) {
        printf("Error3: %s\n", hipGetErrorString(err));
    }
    err = hipMemcpy(d_min, &min_value, sizeof(int), hipMemcpyHostToDevice);
    if(hipGetLastError() != hipSuccess) {
        printf("Error4: %s\n", hipGetErrorString(err));
    }
    err = hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);
    if(hipGetLastError() != hipSuccess) {
        printf("Error5: %s\n", hipGetErrorString(err));
    }
    end_mem = clock();
    double time_taken_mem = double(end_mem - start_mem) / double(CLOCKS_PER_SEC);
    std::cout << "Time taken for memory allocation: \n" << time_taken_mem << std::endl;

    // ----------------- FIXPOINT MODEL ----------------- //
    time_t start, end;
    start = clock();
    // loop until min_value converges to a fixpoint => does not change between iterations
    while(min_value < prev_min_value) {
        // update the previous value
        prev_min_value = min_value;
        findMinFixpointKernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_arr, d_size, d_min);
        if(hipGetLastError() != hipSuccess) {
            printf("Fixpoint kernel Error: %s\n", hipGetErrorString(err));
        }
        // copy the result back to the host for comparison
        err = hipMemcpy(&min_value, d_min, sizeof(int), hipMemcpyDeviceToHost);
        if(hipGetLastError() != hipSuccess) {
            printf("Memcpy Error: %s\n", hipGetErrorString(err));
        }
        iters++;
    }
    end = clock();
    double time_taken = double(end - start) / double(CLOCKS_PER_SEC);
    std::cout << "Time taken by fixpoint iteration: " << time_taken << std::endl;
    std::cout << "Number of fixpoint iterations: " << iters << std::endl;
    printf("Fixpoint computed minimum value is %d\n", min_value);

    // ----------------- PARALLEL REDUCTION KERNEL ----------------- //
    // measure time
    start = clock();
    // reset min_value
    min_value = 1000;
    err = hipMemcpy(d_min, &min_value, sizeof(int), hipMemcpyHostToDevice);
    if(hipGetLastError() != hipSuccess) {
        printf("Error6: %s\n", hipGetErrorString(err));
    }
    findMinKernel<<<BLOCK_SIZE, BLOCK_SIZE>>>(d_arr, d_size, d_min);
    if(hipGetLastError() != hipSuccess) {
        printf("Reduction kernel Error: %s\n", hipGetErrorString(err));
    }
    // copy the result back to the host
    err = hipMemcpy(&min_value, d_min, sizeof(int), hipMemcpyDeviceToHost);
    if(hipGetLastError() != hipSuccess) {
        printf("Memcpy Error: %s\n", hipGetErrorString(err));
    }
    end = clock();
    time_taken = double(end - start) / double(CLOCKS_PER_SEC);

    // copy the result back to the host
    hipMemcpy(&min_value, d_min, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
    hipFree(d_min);
    hipFree(d_size);

    std::cout << "Time taken by parallel reduction: " << time_taken << std::endl;
    std::cout << "Parallel reduction computed minimum value is " << min_value << std::endl;
}

// serial CPU function for comparison
int findMin(std::vector<int> arr) {
    int min = arr[0];
    for (int i = 1; i < arr.size(); i++) {
        if (arr[i] < min) {
            min = arr[i];
        }
    }
    return min;
}

int main(int argc, char **argv) {
    // read the size of the array from the command line
    if (argc != 2) {
        std::cout << "usage: " << argv[0] << " <input dimension> " << std::endl;
        exit(1);
    }
    int n = atoi(argv[1]);

    // generate array of random integers of size n
    // Initialize a random number generator
    int min = 2;
    int max = 1000;
    // initialize random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> distrib(min, max);
    std::vector<int> arr(n);
    
    // if openmp is enabled, set the number of threads
    # ifdef _OPENMP
    omp_set_num_threads(32);
    // print number of active omp threads
    std::cout << "Number of active threads: " << omp_get_max_threads() << std::endl;
    # endif

    # pragma omp parallel for
    for (int i = 0; i < n; i++) {
        arr[i] = distrib(gen);
    }
    // allocate memory on the devide
    kernel_wrapper(arr);

    time_t start, end;
    start = clock();
    int min_value_cpu = findMin(arr);
    end = clock();
    double time_taken = double(end - start) / double(CLOCKS_PER_SEC);
    std::cout << "Time taken CPU: " << time_taken << std::endl;
    printf("CPU computed minimum value is %d\n", min_value_cpu);
    return 0;
}